#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<algorithm>
#include<vector>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<limits>
#include<iomanip>

using namespace std;

#define BLOCK_SIZE 8


__global__ void kernel(int *dim,float *k_k, int *Xlocation, int *Ylocation, int *Zlocation, int *Width, int *Height, int *Depth, float *FixedTemperature, float *OldTemp, float *NewTemp, int *Xaxis, int *Yaxis, int *Zaxis, int *HeatSources){



    if (*dim ==50){

        __shared__ float temp[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
        int j = blockIdx.x*blockDim.x + threadIdx.x;
        int i = blockIdx.y*blockDim.y + threadIdx.y;
        int leni, lenj;

        if (i <= *Yaxis && j <= *Xaxis){

            int threadj = threadIdx.x + 1;
            int threadi = threadIdx.y + 1;

            temp[threadi][threadj] = OldTemp[j+i*(*Yaxis)];

            if (blockIdx.x==(int)*Xaxis/BLOCK_SIZE){

                lenj = *Xaxis%BLOCK_SIZE;


            }
            else{

                lenj = BLOCK_SIZE;
            }

            if (blockIdx.y ==(int)*Yaxis/BLOCK_SIZE){

                leni = *Yaxis%BLOCK_SIZE;
            }
            else{

                leni = BLOCK_SIZE;
            }



            if (threadIdx.x < 1){

                if (j < 1){

                    temp[threadi][threadj - 1] = *(OldTemp + i*(*Yaxis) + j);
                }
                else{
                    temp[threadi][threadj - 1] = *(OldTemp + i*(*Yaxis) + j - 1);
                }

                if (j >= *Xaxis - lenj){
                    temp[threadi][threadj + lenj] = *(OldTemp + i*(*Yaxis)+j +lenj-1);


                }
                else{
                    temp[threadi][threadj + lenj] = *(OldTemp + i*(*Yaxis)+j +lenj);
                }


            }


            if (threadIdx.y < 1 ){

                if(i<1){

                    temp[threadi - 1][threadj] = *(OldTemp + i*(*Yaxis)+j);

                }
                else{

                    temp[threadi - 1][threadj] = *(OldTemp + (i-1)*(*Yaxis)+j);

                }

                if(i >= (*Yaxis) - leni){

                    temp[threadi+leni][threadj] = *(OldTemp+(i+leni-1)*(*Yaxis)+j);


                }
                else{

                    temp[threadi+leni][threadj] = *(OldTemp+(i+leni)*(*Yaxis)+j);
                }

            }

            __syncthreads();

        if (i < *Yaxis && j < *Xaxis){

            NewTemp[j+i*(*Yaxis)] = temp[threadi][threadj] + *k_k*(temp[threadi-1][threadj] + temp[threadi][threadj -1] + temp[threadi+1][threadj] + temp[threadi][threadj+1] - 4*temp[threadi][threadj]);



            for (int p = 0; p < *HeatSources; p++){

                if((i > Ylocation[p]-1) && (i <= Ylocation[p] + Height[p]-1) && (j > Xlocation[p]-1) && (j <= Xlocation[p] + Width[p]-1)){

                    NewTemp[j+(i)*(*Yaxis)] = FixedTemperature[p];

                }


            }

            OldTemp[j+i*(*Yaxis)] = NewTemp[j+i*(*Yaxis)];


        }

        }
    }

    else if (*dim == 51){

        __shared__ float temp[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2];
        int j = blockIdx.x*blockDim.x + threadIdx.x;
        int i = blockIdx.y*blockDim.y + threadIdx.y;
        int k = blockIdx.z*blockDim.z + threadIdx.z;
        int leni, lenj, lenk;

        if (i <= *Yaxis && j <= *Xaxis && k <= *Zaxis){


            int threadj = threadIdx.x + 1;
            int threadi = threadIdx.y + 1;
            int threadk = threadIdx.z + 1;

            temp[threadi][threadj][threadk] = OldTemp[(j+i*(*Yaxis))*(*Zaxis) + k];

            if (blockIdx.x==(int)*Xaxis/BLOCK_SIZE){

                lenj = *Xaxis%BLOCK_SIZE;


            }
            else{

                lenj = BLOCK_SIZE;
            }

            if (blockIdx.y ==(int)*Yaxis/BLOCK_SIZE){

                leni = *Yaxis%BLOCK_SIZE;
            }
            else{

                leni = BLOCK_SIZE;
            }

            if (blockIdx.z == (int)*Zaxis/BLOCK_SIZE){
                lenk = *Zaxis%BLOCK_SIZE;
            }
            else{
                lenk = BLOCK_SIZE;
            }



            if (threadIdx.x < 1){

                if (j < 1){

                    temp[threadi][threadj - 1][threadk] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k);
                }
                else{
                    temp[threadi][threadj - 1][threadk] = *(OldTemp + (j-1+i*(*Yaxis))*(*Zaxis) + k);
                }

                if (j >= *Xaxis - lenj){
                    temp[threadi][threadj + lenj][threadk] = *(OldTemp + (j+lenj - 1 + i*(*Yaxis))*(*Zaxis) + k);


                }
                else{
                    temp[threadi][threadj + lenj][threadk] = *(OldTemp + (j+lenj + i*(*Yaxis))*(*Zaxis) + k);
                }


            }



            if (threadIdx.y < 1 ){

                if(i<1){

                    temp[threadi - 1][threadj][threadk] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k);

                }
                else{

                    temp[threadi - 1][threadj][threadk] = *(OldTemp + (j+(i-1)*(*Yaxis))*(*Zaxis) + k);

                }

                if(i >= (*Yaxis) - leni){

                    temp[threadi+leni][threadj][threadk] = *(OldTemp + (j+(i + leni-1)*(*Yaxis))*(*Zaxis) + k);


                }
                else{

                    temp[threadi+leni][threadj][threadk] = *(OldTemp + (j+(i + leni)*(*Yaxis))*(*Zaxis) + k);
                }

            }

            if (threadIdx.z < 1 ){

                if(k<1){

                    temp[threadi][threadj][threadk-1] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k);
                }
                else{

                    temp[threadi][threadj][threadk-1] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k - 1);

                }

                if(k >= (*Zaxis) - lenk){

                    temp[threadi][threadj][threadk + lenk] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k + lenk - 1);


                }
                else{

                    temp[threadi][threadj][threadk + lenk] = *(OldTemp + (j+i*(*Yaxis))*(*Zaxis) + k + lenk);
                }

            }

            __syncthreads();

            if (i < *Yaxis && j < *Xaxis && k<*Zaxis){

                NewTemp[(j+i*(*Yaxis))*(*Zaxis) + k] = temp[threadi][threadj][threadk] + *k_k*(temp[threadi-1][threadj][threadk] + temp[threadi][threadj -1][threadk] + temp[threadi+1][threadj][threadk] + temp[threadi][threadj+1][threadk]+temp[threadi][threadj][threadk-1]+temp[threadi][threadj][threadk+1] - 6*temp[threadi][threadj][threadk]);

                for (int p = 0; p < *HeatSources; p++){


                    if((i > Ylocation[p]-1) && (i <= Ylocation[p] + Height[p]-1) && (j > Xlocation[p]-1) && (j <= Xlocation[p] + Width[p]-1)&& (k >= Zlocation[p])&& (k < Zlocation[p] + Depth[p])){

                        NewTemp[(j+i*(*Yaxis))*(*Zaxis) + k] = FixedTemperature[p];

                    }

                }

                OldTemp[(j+i*(*Yaxis))*(*Zaxis) + k] = NewTemp[(j+i*(*Yaxis))*(*Zaxis) + k];
            }
        }

    }

}

int main(int argc, char const *argv[])
{
    int StringLength = 0;

    ifstream inFile(argv[1]);
    string strOneLine;
    int ParaNumber= 0;
    int dim = 0;
    vector <string> parastring;
    float k_k, StartTemp, FixedTemperature[26];
    int TimeSteps, Xaxis, Yaxis, Zaxis, Xlocation[26], Ylocation[26], Zlocation[26], Width[26], Height[26], Depth[26];
    int HeatSources;
    string CurrString;
    int len;

    while(inFile)
    {

        getline(inFile, strOneLine);
        StringLength = strOneLine.length();
        if (StringLength==0|| int(strOneLine[0]) == 13) continue;



        for(int i = 0; i<StringLength; i ++){

            if (strOneLine.at(i) == ' ') continue;

            if (strOneLine.at(i) == '#')
            {
                break;
            }

            if (strOneLine.at(i) == ',') {

                ParaNumber = ParaNumber + 1;
                parastring.push_back(CurrString);
                CurrString.clear();
                continue;
            }

            if (ParaNumber == 0){

                dim = int(strOneLine.at(i));
                ParaNumber=ParaNumber+1;
                break;

            }


            else
            {

                CurrString.push_back(strOneLine.at(i));
           }

        }
        if(CurrString.length()!= 0){

            parastring.push_back(CurrString);
            CurrString.clear();
            ParaNumber = ParaNumber + 1;

        }


    }

    int *d_Xlocation, *d_Ylocation, *d_Zlocation, *d_Width, *d_Height, *d_Depth, *d_Xaxis, *d_Yaxis, *d_Zaxis, *d_dim, *d_HeatSources;
    float *d_OldTemp, *d_NewTemp, *d_k, *d_FixedTemperature;

    if (dim==50)
    {

        parastring[0].insert(0,1,'0');
        k_k = atof(parastring[0].c_str());
        TimeSteps = atoi(parastring[1].c_str());
        Xaxis = atoi(parastring[2].c_str());
        Yaxis = atoi(parastring[3].c_str());
        StartTemp = atof(parastring[4].c_str());
        HeatSources = (ParaNumber-5)/5;

        for (int i=5;i<ParaNumber-5;i=i+5){

            Xlocation[(i-5)/5] = atoi(parastring[i].c_str());
            Ylocation[(i-5)/5] = atoi(parastring[i+1].c_str());
            Width[(i-5)/5] = atoi(parastring[i+2].c_str());
            Height[(i-5)/5] = atoi(parastring[i+3].c_str());
            FixedTemperature[(i-5)/5] = atof(parastring[i+4].c_str());


        }
        //cout<<"Dimension = "<<dim<<endl;
        //cout<<"k = "<<k_k<<endl;
        //cout<<"Time Steps = "<<TimeSteps<<endl;
        //cout<<"X Axis, Y axis = "<<Xaxis<<", "<<Yaxis<<endl;
        //cout<<"Starting Temp = "<<StartTemp<<endl;

        //for (int i=0; i<HeatSources; i++)
        //{
        //    cout<<Xlocation[i]<<", "<<Ylocation[i]<<", "<<Width[i]<<", "<<Height[i]<<", "<<FixedTemperature[i]<<endl;
        //}

        if (Xaxis >= Yaxis){

            len = Xaxis*Xaxis;
        }
        else{
            len = Yaxis*Yaxis;
        }

        int size = (len)*sizeof(float);

        float OldTemp[len] = {0};
        float NewTemp[len] = {0};

        for (int i = 0; i < Yaxis ; i ++){
            for (int j = 0; j < Xaxis; j ++){
                OldTemp[i*Yaxis + j] = StartTemp;

                for (int p = 0; p < HeatSources; p++){

                    if((i > Ylocation[p]-1) && (i <= Ylocation[p] + Height[p]-1) && (j > Xlocation[p]-1) && (j <= Xlocation[p] + Width[p]-1)){

                        OldTemp[i*Yaxis + j] = FixedTemperature[p];

                    }
                }
            }
        }

        hipMalloc((void **)&d_OldTemp, size);
        hipMalloc((void **)&d_NewTemp, size);
        hipMalloc((void **)&d_Xlocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Ylocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Xaxis, sizeof(int));
        hipMalloc((void **)&d_Yaxis, sizeof(int));
        hipMalloc((void **)&d_k, sizeof(float));
        hipMalloc((void **)&d_Width, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Height, HeatSources*sizeof(int));
        hipMalloc((void **)&d_FixedTemperature, HeatSources*sizeof(float));
        hipMalloc((void **)&d_Zlocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Depth, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Zaxis, sizeof(int));
        hipMalloc((void **)&d_dim, sizeof(int));
        hipMalloc((void **)&d_HeatSources, sizeof(int));

        hipMemcpy(d_OldTemp, OldTemp, size, hipMemcpyHostToDevice);
        hipMemcpy(d_NewTemp, NewTemp, size, hipMemcpyHostToDevice);
        hipMemcpy(d_Xlocation, Xlocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Ylocation, Ylocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Xaxis, &Xaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Yaxis, &Yaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_k, &k_k, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Width, Width, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Height, Height, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_FixedTemperature, FixedTemperature, HeatSources*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Zlocation, Zlocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Zaxis, &Zaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Depth, Depth, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_HeatSources, &HeatSources, sizeof(int), hipMemcpyHostToDevice);

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((int)ceil(((Xaxis + BLOCK_SIZE - 1)/BLOCK_SIZE)),(int)(ceil(((Yaxis + BLOCK_SIZE - 1)/BLOCK_SIZE))));

        for (int i = 0; i < TimeSteps; i++){

            kernel<<<grid,block>>>(d_dim, d_k, d_Xlocation, d_Ylocation, d_Zlocation, d_Width, d_Height, d_Depth, d_FixedTemperature, d_OldTemp, d_NewTemp, d_Xaxis, d_Yaxis, d_Zaxis, d_HeatSources);

        }
        hipMemcpy(NewTemp, d_NewTemp, size, hipMemcpyDeviceToHost);

        hipFree(d_OldTemp);
        hipFree(d_NewTemp);
        hipFree(d_Xlocation);
        hipFree(d_Ylocation);
        hipFree(d_Xaxis);
        hipFree(d_Yaxis);
        hipFree(d_k);
        hipFree(d_Width);
        hipFree(d_Height);
        hipFree(d_FixedTemperature);
        hipFree(d_Zlocation);
        hipFree(d_Zaxis);
        hipFree(d_Depth);
        hipFree(d_dim);
        hipFree(d_HeatSources);


        ofstream build ("heatOutput.csv", std::ofstream::out);
        for(int i = 0; i <Yaxis; i++){
            for(int j = 0; j <Xaxis - 1; j++){

                build<<NewTemp[i*Yaxis + j]<<", ";

            }
            build<<NewTemp[i*Yaxis + Xaxis-1];
            if(i != Yaxis -1){

                build<<endl;

            }
        }
        build.close();

    }

    if (dim ==51)
    {
        parastring[0].insert(0,1,'0');
        k_k = atof(parastring[0].c_str());
        TimeSteps = atoi(parastring[1].c_str());
        Xaxis = atoi(parastring[2].c_str());
        Yaxis = atoi(parastring[3].c_str());
        Zaxis = atoi(parastring[4].c_str());
        StartTemp = atof(parastring[5].c_str());
        HeatSources = (ParaNumber-7)/5;


        for (int i=6;i<ParaNumber-6;i=i+7){

            Xlocation[(i-6)/7] = atoi(parastring[i].c_str());
            Ylocation[(i-6)/7] = atoi(parastring[i+1].c_str());
            Zlocation[(i-6)/7] = atoi(parastring[i+2].c_str());
            Width[(i-6)/7] = atoi(parastring[i+3].c_str());
            Height[(i-6)/7] = atoi(parastring[i+4].c_str());
            Depth[(i-6)/7] = atoi(parastring[i+5].c_str());
            FixedTemperature[(i-6)/7] = atof(parastring[i+6].c_str());

        }

        //cout<<"Dimension = "<<dim<<endl;
        //cout<<"k = "<<k_k<<endl;
        //cout<<"Time Steps = "<<TimeSteps<<endl;
        //cout<<"X Axis, Y axis, Z axis = "<<Xaxis<<", "<<Yaxis<<", "<<Zaxis<<endl;
        //cout<<"Starting Temp = "<<StartTemp<<endl;
        //cout<<"HeatSources = "<<HeatSources<<endl;

        //for (int i=0; i<HeatSources; i++)
        //{
          //  cout<<Xlocation[i]<<", "<<Ylocation[i]<<", "<<Zlocation[i]<<", "<<Width[i]<<", "<<Height[i]<<", "<<Depth[i]<<", "<<FixedTemperature[i]<<endl;
        //}

        if (Xaxis >= Yaxis && Xaxis >= Zaxis){

            len = Xaxis*Xaxis*Xaxis;
        }
        else if (Yaxis >= Xaxis && Yaxis >= Zaxis){
            len = Yaxis*Yaxis*Yaxis;
        }
        else if (Zaxis>= Yaxis &&  Zaxis>=Yaxis){

            len = Zaxis*Zaxis*Zaxis;
        }

        int size = (len)*sizeof(float);

        float OldTemp[len] = {0};
        float NewTemp[len] = {0};

        for(int k = 0; k <Zaxis; k++){

            for (int i = 0; i < Yaxis ; i ++){
                for (int j = 0; j < Xaxis; j ++){
                    OldTemp[(j+i*(Yaxis))*(Zaxis) + k] = StartTemp;

                    for (int p = 0; p < HeatSources; p++){

                        if((i > Ylocation[p]-1) && (i <= Ylocation[p] + Height[p]-1) && (j > Xlocation[p]-1) && (j <= Xlocation[p] + Width[p]-1) && (k > Zlocation[p] - 1) && (k<= Zlocation[p] + Depth[p] - 1)){

                            OldTemp[(j+i*(Yaxis))*(Zaxis) + k] = FixedTemperature[p];

                        }
                    }
                }
            }
        }

        hipMalloc((void **)&d_OldTemp, size);
        hipMalloc((void **)&d_NewTemp, size);
        hipMalloc((void **)&d_Xlocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Ylocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Xaxis, sizeof(int));
        hipMalloc((void **)&d_Yaxis, sizeof(int));
        hipMalloc((void **)&d_k, sizeof(float));
        hipMalloc((void **)&d_Width, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Height, HeatSources*sizeof(int));
        hipMalloc((void **)&d_FixedTemperature, HeatSources*sizeof(float));
        hipMalloc((void **)&d_Zlocation, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Depth, HeatSources*sizeof(int));
        hipMalloc((void **)&d_Zaxis, sizeof(int));
        hipMalloc((void **)&d_dim, sizeof(int));
        hipMalloc((void **)&d_HeatSources, sizeof(int));

        hipMemcpy(d_OldTemp, OldTemp, size, hipMemcpyHostToDevice);
        hipMemcpy(d_NewTemp, NewTemp, size, hipMemcpyHostToDevice);
        hipMemcpy(d_Xlocation, Xlocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Ylocation, Ylocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Xaxis, &Xaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Yaxis, &Yaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_k, &k_k, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Width, Width, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Height, Height, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_FixedTemperature, FixedTemperature, HeatSources*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Zlocation, Zlocation, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Zaxis, &Zaxis, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_Depth, Depth, HeatSources*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_HeatSources, &HeatSources, sizeof(int), hipMemcpyHostToDevice);

        dim3 block(BLOCK_SIZE, BLOCK_SIZE,BLOCK_SIZE);
        dim3 grid((int)ceil(((Xaxis + BLOCK_SIZE - 1)/BLOCK_SIZE)),(int)(ceil(((Yaxis + BLOCK_SIZE - 1)/BLOCK_SIZE))),(int)(ceil(((Zaxis + BLOCK_SIZE - 1)/BLOCK_SIZE))));
        for (int i = 0; i < TimeSteps; i++){

            kernel<<<grid,block>>>(d_dim, d_k, d_Xlocation, d_Ylocation, d_Zlocation, d_Width, d_Height, d_Depth, d_FixedTemperature, d_OldTemp, d_NewTemp, d_Xaxis, d_Yaxis, d_Zaxis, d_HeatSources);

        }
        hipMemcpy(NewTemp, d_NewTemp, size, hipMemcpyDeviceToHost);

        hipFree(d_OldTemp);
        hipFree(d_NewTemp);
        hipFree(d_Xlocation);
        hipFree(d_Ylocation);
        hipFree(d_Xaxis);
        hipFree(d_Yaxis);
        hipFree(d_k);
        hipFree(d_Width);
        hipFree(d_Height);
        hipFree(d_FixedTemperature);
        hipFree(d_Zlocation);
        hipFree(d_Zaxis);
        hipFree(d_Depth);
        hipFree(d_dim);
        hipFree(d_HeatSources);


        ofstream build ("heatOutput.csv", std::ofstream::out);
        for (int l = 0; l < Zaxis; l++){
            for(int i = 0; i <Yaxis; i++){
                for(int j = 0; j <Xaxis - 1; j++){

                    build<<NewTemp[(j+i*(Yaxis))*(Zaxis) + l]<<", ";

                }
                build<<NewTemp[(Xaxis - 1 +i*(Yaxis))*(Zaxis) + l];

                if(l != Zaxis -1 || i != Yaxis -1){

                    build<<endl;
                }
            }

            if (l != Zaxis - 1){
                build<<endl;

            }

        }
        build.close();

    }

    return 0;
}






